#include "hip/hip_runtime.h"
__global__ void cuda_reduction_5(I *z, I *x, I bound){
    extern __shared__ I sharedData[];
    UI tid = threadIdx.x;
    UI i = blockIdx.x * blockDim.x + threadIdx.x;
    sharedData[tid] = (i<bound)?x[i]:0;
    __syncthreads(); // load to shared memory
    //printf("tid.x = %d, i = %d, dim = %d\n", threadIdx.x,(I)i,blockDim.x);
    // reduction
    for(UI k=blockDim.x>>2; k>0; k>>=2){ // require 4^x = blockDim.x
        if(tid < k){
            //P("1 k = %d, tid = %d, a = %d, b = %d, c = %d, d = %d\n", k,tid,\
                    sharedData[tid], sharedData[tid + k], sharedData[tid + k*2],sharedData[tid + k*3]);
            //I xx = sharedData[tid + k] + sharedData[tid + (k*2)] + sharedData[tid + k * 3];
            sharedData[tid] += sharedData[tid + k] + sharedData[tid + (k<<1)] + sharedData[tid + (k<<1) + k];
            //P("2 k = %d, tid = %d, a = %d, b = %d, c = %d, d = %d, xx = %d\n", k,tid,\
                    sharedData[tid], sharedData[tid + k], sharedData[tid + k*2],sharedData[tid + k*3], xx);
        }
        __syncthreads(); 
    }
    //P("blockIdx = %d, tid = %d, i = %d, sharedData[%d] = %d\n", \
            blockIdx.x, tid, (I)i, tid, sharedData[tid]);
    // write result to global mem
    if(tid == 0) z[blockIdx.x] = sharedData[0];
}

static I run_gpu_v5(I *x, L n){
    I *dx, *dz, *z;
    I numThread = NUM_THREAD;
    I numBlock  = (n/numThread) + (n%numThread!=0);
    I memSize   = numThread * sizeof(I);
    NEW(z, I, numBlock);
    CUDA_NEW(dx, I, n);
    CUDA_NEW(dz, I, numBlock);
    // copy
    CUDA_COPY(dx, x, I, n);
    //CUDA_COPY(dz, x, F, numBlock);
    //DOI(15, P("x[%lld] = %f\n", i,x[i]))
    struct timeval t1, t2;
    P("# of Block: %d\n# of Thread: %d\n", numBlock, numThread);
    gettimeofday(&t1, 0);
    // Dynamic Shared Memory
    //   https://devblogs.nvidia.com/using-shared-memory-cuda-cc/
    cuda_reduction_5<<<numBlock, numThread, memSize>>>(dz, dx, n);
    hipDeviceSynchronize(); /* Wait for compute device to finish */
    CUDA_SAVE(z, dz, F, numBlock);
    I rtn = calcSum(z, numBlock);
    gettimeofday(&t2, 0);
    P("[GPU] The elapsed average time (ms): %g (v5)\n", calcTime(t1,t2));
    DOI(5, P("z[%lld] = %d\n", i,z[i]))
    CUDA_FREE(dx);
    CUDA_FREE(dz);
    R rtn;
}


